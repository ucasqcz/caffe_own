#include <vector>
#include "caffe/layers/diversity_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe{

template<typename Dtype>
void DiversityLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top){
	int n = bottom[0]->shape()[0];	//��������
	int b_num = bottom.size();		//branch����
	int d = bottom[0]->count();

	Dtype loss = 0.0;
	for (int i = 0; i < b_num - 1; i++){
		for (int j = i + 1; j < b_num; j++){
			Blob<Dtype> diff(bottom[0]->shape());
			caffe_gpu_sub(d, bottom[i]->gpu_data(), bottom[j]->gpu_data(), diff.mutable_gpu_data());
			//caffe_gpu_axpby(d, Dtype(1), diff.gpu_data(), Dtype(1), diff_.mutable_gpu_data() + i*d);
			//caffe_gpu_axpby(d, Dtype(-1), diff.gpu_data(), Dtype(1), diff_.mutable_gpu_data() + j*d);
			caffe_gpu_add(d, diff.gpu_data(), diff_.gpu_data() + i*d, diff_.mutable_gpu_data() + i*d);
			caffe_gpu_sub(d, diff_.gpu_data() + j*d, diff.gpu_data(), diff_.mutable_gpu_data() + j*d);
			caffe_gpu_powx(d, diff.gpu_data(), Dtype(2), diff.mutable_gpu_data());
			Dtype dot = 0.0;
			caffe_gpu_asum(d, diff.gpu_data(),&dot);
			loss += dot;
		}
	}
	if (b_num != 1){
		loss = Dtype(1) - loss / Dtype(b_num * (b_num - 1) * n);
	}
	top[0]->mutable_cpu_data()[0] = loss;
	//caffe_gpu_set(1, Dtype(loss), top[0]->mutable_gpu_data());
}

template<typename Dtype>
void DiversityLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){
	int n = bottom[0]->num();	//��������
	int b_num = bottom.size();		//branch����
	const int d = bottom[0]->count();
	const Dtype* diff = diff_.gpu_data();
	if (b_num != 1){
		const Dtype alpha = Dtype(-1) * Dtype(2)* top[0]->cpu_diff()[0] / Dtype(b_num * (b_num - 1)* n);
		for (int i = 0; i < b_num; i++){
			if (propagate_down[i]){
				caffe_gpu_scale(d,alpha,diff,bottom[i]->mutable_gpu_diff());
				diff += d;
			}
		}
	}

}
INSTANTIATE_LAYER_GPU_FUNCS(DiversityLossLayer);
}
